
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add_a(int *A, int *B, int *C, int N) {
    int i = threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 5;
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    h_A = (int*)malloc(N * sizeof(int));
    h_B = (int*)malloc(N * sizeof(int));
    h_C = (int*)malloc(N * sizeof(int));

    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    hipMalloc(&d_A, N * sizeof(int));
    hipMalloc(&d_B, N * sizeof(int));
    hipMalloc(&d_C, N * sizeof(int));

    hipMemcpy(d_A, h_A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(int), hipMemcpyHostToDevice);

    // 1 block of N threads
    add_a<<<1, N>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        printf("%d ", h_C[i]);
    }
    printf("\naddition completed successfully.\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
