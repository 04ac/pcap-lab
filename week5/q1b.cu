#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void add_b(int* c, int* a, int* b, int N) {
	int i = blockIdx.x;

	if (i < N) {
		c[i] = a[i] + b[i];
	}
}

int main() {
	int* a, *b, *c;
	int* d_a, *d_b, *d_c;
	int N = 5;

	// allocate host memory
	a = (int*) malloc(N * sizeof(int));
	b = (int*) malloc(N * sizeof(int));
	c = (int*) malloc(N * sizeof(int));

	// Initialize vectors
	for (int i = 0; i < n; i++) {
		a[i] = i;
		b[i] = 2 * i;
	}

	// allocate device memory
	hipMalloc(&d_a, N * sizeof(int));
	hipMalloc(&d_b, N * sizeof(int));
	hipMalloc(&d_c, N * sizeof(int));

	hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	add_b<<<N, 1>>>(d_c, d_a, d_b, N);

	hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        printf("%d ", c[i]);
    }
    printf("\naddition completed successfully.\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}