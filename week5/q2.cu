
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void addVecs(int* c, int* a, int* b, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n) {
		c[i] = a[i] + b[i];
	}
}

__global__ void init_d(int* d, int n, int factor) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n) {
		d[i] = factor * i;
	}
}

int main() {
	int *c;
	int* d_a, *d_b, *d_c;

	int THREADS_PER_BLOCK = 256;
	int n = 257;

	c = (int*) malloc(n * sizeof(int));

	hipMalloc(&d_a, n * sizeof(int));
	hipMalloc(&d_b, n * sizeof(int));
	hipMalloc(&d_c, n * sizeof(int));

	int NUM_BLOCKS = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK; // for ceil function

	// Initialize device memory
	init_d<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_a, n, 1);
	init_d<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_b, n, 2);

	addVecs<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_c, d_a, d_b, n);

	hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < n; i++) {
		printf("%d\t", c[i]);
	}
	printf("\nAddition Completed\n");

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	free(c);

	return 0;
}