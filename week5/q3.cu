
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void sinVec(double* res, double* inp, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n) {
		res[i] = sin(inp[i]);
	}
}

__global__ void initVec(double* inp, int n) {
	// vector is initialized with multiples of pi / 6

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < n) {
		inp[i] = i * (M_PI / 6);
	}
}

int main() {
	double* inp, *res, *d_res;

	int THREADS_PER_BLOCK = 4;
	int n = 15;

	hipMalloc(&inp, n * sizeof(double));
	hipMalloc(&d_res, n *  sizeof(double));
	res = (double*) malloc(n * sizeof(double));

	int numBlocks = ceil((double) n / THREADS_PER_BLOCK);

	initVec<<<numBlocks, THREADS_PER_BLOCK>>>(inp, n);
	sinVec<<<numBlocks, THREADS_PER_BLOCK>>>(d_res, inp, n);

	hipMemcpy(res, d_res, n * sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < n; i++) {
		printf("%.5f\t", res[i]);
	}
	printf("\n");

	hipFree(d_res);
	hipFree(inp);
	free(res);
	return 0;
}