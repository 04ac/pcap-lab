#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void conv1D(float* n, float* m, float* p, int width, int maskWidth) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < width) {
        float k = 0;

        int nStartVal = i - (maskWidth / 2);

        for (int j = 0; j < maskWidth; j++) {
            int nIdx = j + nStartVal;

            if (nIdx >= 0 && nIdx < width) {
                k += n[nIdx] * m[j];
            }
        }
        p[i] = k;
    }
}

int main() {
    int n1,n2;

    printf("Length of the vector: ");
    scanf("%d", &n1);

    printf("Enter the length of mask: ");
    scanf("%d", &n2);

    float n[n1], m[n2], p[n1];
    float *dn, *dm, *dp;

    hipMalloc((void **) &dn, n1 * sizeof(float));
    hipMalloc((void **) &dm, n2 * sizeof(float));
    hipMalloc((void **) &dp, n1 * sizeof(float));

    printf("Enter vector: ");
    for(int i = 0; i < n1; i++)
        scanf("%f", &n[i]);

    printf("Enter mask: ");
    for(int i = 0; i < n2; i++)
        scanf("%f", &m[i]);

    hipMemcpy(dn, n, n1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dm, m, n2 * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(n1,1,1);
    dim3 blk(1,1,1);

    conv1D<<<grid,blk>>>(dn, dm, dp, n1, n2);
    hipMemcpy(p, dp, n1 * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < n1; i++)
        printf("%f\t", p[i]);
    printf("\n");

    hipFree(dm);
    hipFree(dn);
    hipFree(dp);
}
