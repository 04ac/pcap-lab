#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define MAX_W 1024
#define MAX_LEN 50

__device__ int gpu_strcmp(char *s1, char *s2) {
    while (*s1 && *s2 && (*s1 == *s2)) {
        s1++;
        s2++;
    }
    return (*s1 - *s2);  // Return difference of mismatched characters
}

__global__ void countWords(char* d_words, char* d_target, int* d_cnt, int numWords) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= numWords) return;

	char* curr = &d_words[i * MAX_LEN];

	if (gpu_strcmp(curr, d_target) == 0) {
		atomicAdd(d_cnt, 1);
	}
}

int main() {
	char h_sentence[] = "hello how is life, life is good!";
	char h_target[] = "life";
	int h_cnt = 0;

	char h_words[MAX_W][MAX_LEN];
	int numWords = 0;

	char* token = strtok(h_sentence, " ,!?."); // split on spaces and punctuation

	while (token != NULL && numWords < MAX_W) {
		strcpy(h_words[numWords++], token);
		token = strtok(NULL, " ,!?.");
	}

	char* d_words, *d_target;
	int* d_cnt;

	hipMalloc((void**) &d_words, numWords * MAX_LEN);
	hipMalloc((void**) &d_target, strlen(h_target) + 1);
	hipMalloc((void**) &d_cnt, sizeof(int));

	hipMemcpy(d_words, h_words, numWords * MAX_LEN, hipMemcpyHostToDevice);
	hipMemcpy(d_target, h_target, strlen(h_target) + 1, hipMemcpyHostToDevice);
	hipMemcpy(d_cnt, &h_cnt, sizeof(int), hipMemcpyHostToDevice);

	int numThreads = 256;
	int numBlocks = ceil((1.0 * numWords) / numThreads);

	countWords<<<numBlocks, numThreads>>>(d_words, d_target, d_cnt, numWords);

	hipMemcpy(&h_cnt, d_cnt, sizeof(int), hipMemcpyDeviceToHost);

	printf("The word \"%s\" appears %d times\n", h_target, h_cnt);

	hipFree(d_words);
	hipFree(d_target);
	hipFree(d_cnt);
}
