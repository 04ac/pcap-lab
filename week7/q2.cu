#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__device__ int sumUpto(int n) {
    return (n * (n + 1)) / 2;
}

__global__ void makeString(char* d_rs, char* d_s, int len, int rs_len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= len) return;

    int offset = sumUpto(len) - sumUpto(len - i);

    for (int k = offset; k < offset + len - i; k++) {
        d_rs[k] = d_s[k - offset];
    }
}

int main() {
    char h_s[50];

    printf("Enter string S: ");
    scanf("%s", h_s);

    int len = strlen(h_s);

    int rs_len = len * (len + 1) / 2; // 4 + 3 + 2 + 1 for PCAP
    char h_rs[rs_len + 1];

    char* d_s, *d_rs;
    hipMalloc((void**) &d_s, len + 1);
    hipMalloc((void**) &d_rs, rs_len + 1);

    hipMemcpy(d_s, h_s, len + 1, hipMemcpyHostToDevice);

    int numThreads = 256;
    int numBlocks = ceil((1.0 * len) / numThreads);

    makeString<<<numBlocks, numThreads>>>(d_rs, d_s, len, rs_len);

    hipMemcpy(h_rs, d_rs, rs_len + 1, hipMemcpyDeviceToHost);
    h_rs[rs_len] = '\0';

    printf("String RS: %s\n", h_rs);

    hipFree(d_s);
    hipFree(d_rs);
}