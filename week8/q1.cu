#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matrixAddRowWise(int* A, int* B, int* C, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < rows) {
        int rowOffset = row * cols;
        for (int j = 0; j < cols; j++) {
            C[rowOffset + j] = A[rowOffset + j] + B[rowOffset + j];
        }
    }
}

__global__ void matrixAddColumnWise(int* A, int* B, int* C, int rows, int cols) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (col < cols) {
        for (int i = 0; i < rows; i++) {
            int index = i * cols + col;
            C[index] = A[index] + B[index];
        }
    }
}

__global__ void matrixAddElementWise(int* A, int* B, int* C, int rows, int cols) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < rows && col < cols) {
        int index = row * cols + col;
        C[index] = A[index] + B[index];
    }
}

// Function to input matrix elements from user
void inputMatrix(int *mat, int rows, int cols, char *name)
{
    printf("Enter the elements of matrix %s (%d x %d):\n", name, rows, cols);
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%s[%d][%d]: ", name, i, j);
            scanf("%d", &mat[i * cols + j]);
        }
    }
}

// Function to print a matrix
void printMatrix(int *mat, int rows, int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%d\t", mat[i * cols + j]);
        }
        printf("\n");
    }
}

int main()
{
    // Matrix dimensions
    int ha, wa, wb;

    printf("Enter rows and cols\n");
    scanf("%d %d", &ha, &wa);

    int hb = ha;
    wb = wa;

    // Check if dimensions are valid for the kernel
    if (ha > 1024)
    {
        printf("Error: Number of rows in matrix A cannot exceed 1024 for this implementation.\n");
        return 1;
    }

    // Host matrices
    int *h_a, *h_b, *h_c;

    // Allocate host memory
    h_a = (int *)malloc(ha * wa * sizeof(int));
    h_b = (int *)malloc(hb * wb * sizeof(int));
    h_c = (int *)malloc(ha * wb * sizeof(int));

    // Get matrix values from user
    inputMatrix(h_a, ha, wa, "A");
    inputMatrix(h_b, hb, wb, "B");

    // Device matrices
    int *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc((void **)&d_a, ha * wa * sizeof(int));
    hipMalloc((void **)&d_b, hb * wb * sizeof(int));
    hipMalloc((void **)&d_c, ha * wb * sizeof(int));

    // Copy host matrices to device
    hipMemcpy(d_a, h_a, ha * wa * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, hb * wb * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16); // 16×16 = 256 threads per block
    dim3 numBlocks(
        (wa + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (ha + threadsPerBlock.y - 1) / threadsPerBlock.y
    );

    // Launch the kernel
    matrixAddElementWise<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, ha, wa);


    hipMemcpy(h_c, d_c, ha * wb * sizeof(int), hipMemcpyDeviceToHost);

    // Print matrices
    printf("\nMatrix A:\n");
    printMatrix(h_a, ha, wa);

    printf("\nMatrix B:\n");
    printMatrix(h_b, hb, wb);

    printf("\nResult Matrix C:\n");
    printMatrix(h_c, ha, wb);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
