#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void multiplyKernel_rowwise(int *a, int *b, int *c, int wa, int wb)
{
    int ridA = threadIdx.x;
    int sum;
    for (int cidB = 0; cidB < wb; cidB++)
    {
        sum = 0;
        for (int k = 0; k < wa; k++)
        {
            sum += (a[ridA * wa + k] * b[k * wb + cidB]);
        }
        c[ridA * wb + cidB] = sum;
    }
}

__global__ void multiplyKernel_colwise(int *a, int *b, int *c, int ha, int wa)
{
    int cidB = threadIdx.x;
    int wb = blockDim.x;
    int sum, k;
    for (int ridA = 0; ridA < ha; ridA++)
    {
        sum = 0;
        for (k = 0; k < wa; k++)
        {
            sum += (a[ridA * wa + k] * b[k * wb + cidB]);
        }
        c[ridA * wb + cidB] = sum;
    }
}

__global__ void multiplyKernel_elementwise(int *a, int *b, int *c, int wa)
{
    int ridA = threadIdx.y;
    int cidB = threadIdx.x;
    int wb = blockDim.x;
    int sum = 0, k;
    for (k = 0; k < wa; k++)
    {
        sum += (a[ridA * wa + k] * b[k * wb + cidB]);
    }
    c[ridA * wb + cidB] = sum;
}

void inputMatrix(int *mat, int rows, int cols, char *name)
{
    printf("Enter the elements of matrix %s (%d x %d):\n", name, rows, cols);
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%s[%d][%d]: ", name, i, j);
            scanf("%d", &mat[i * cols + j]);
        }
    }
}

void printMatrix(int *mat, int rows, int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%d\t", mat[i * cols + j]);
        }
        printf("\n");
    }
}

int main()
{
    // Matrix dimensions
    int ha, wa, wb;

    printf("Enter the dimensions of matrix A (rows columns): ");
    scanf("%d %d", &ha, &wa);

    printf("Enter the number of columns for matrix B: ");
    scanf("%d", &wb);

    // Height of matrix B must equal width of A
    int hb = wa;

    // Host matrices
    int *h_a, *h_b, *h_c;
    
    // Allocate host memory
    h_a = (int *)malloc(ha * wa * sizeof(int));
    h_b = (int *)malloc(hb * wb * sizeof(int));
    h_c = (int *)malloc(ha * wb * sizeof(int));

    // Get matrix values from user
    inputMatrix(h_a, ha, wa, "A");
    inputMatrix(h_b, hb, wb, "B");

    // Device matrices
    int *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc((void **)&d_a, ha * wa * sizeof(int));
    hipMalloc((void **)&d_b, hb * wb * sizeof(int));
    hipMalloc((void **)&d_c, ha * wb * sizeof(int));

    // Copy host matrices to device
    hipMemcpy(d_a, h_a, ha * wa * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, hb * wb * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    multiplyKernel_elementwise<<<(1, 1), (wb,ha)>>>(d_a, d_b, d_c, wa);

    hipMemcpy(h_c, d_c, ha * wb * sizeof(int), hipMemcpyDeviceToHost);

    // Print matrices
    printf("\nMatrix A:\n");
    printMatrix(h_a, ha, wa);

    printf("\nMatrix B:\n");
    printMatrix(h_b, hb, wb);

    printf("\nResult Matrix C (A × B):\n");
    printMatrix(h_c, ha, wb);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
