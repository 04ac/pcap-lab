#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void replace_rows(int *A, int num_rows, int num_cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= num_rows) return;

    for (int j = 0; j < num_cols; j++) {
        int idx = row * num_cols + j;
        int power = row + 1;
        A[idx] = pow(A[idx], power);
    }
}

int main() {
    int M, N;
    printf("Enter no of rows and columns : ");
    scanf("%d %d", &M, &N);

    int *A = (int*)malloc(sizeof(int) * M * N);
    printf("Enter the matrix :\n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) 
            scanf("%d", &A[i * N + j]);
    }

    int *d_A;
    hipMalloc(&d_A, sizeof(int) * M * N);
    hipMemcpy(d_A, A, sizeof(int) * M * N, hipMemcpyHostToDevice);

    replace_rows<<<(M + 255) / 256, 256>>>(d_A, M, N);

    hipMemcpy(A, d_A, sizeof(int) * M * N, hipMemcpyDeviceToHost);

    printf("Modified matrix : \n");
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) 
            printf("%d ", A[i * N + j]);
        printf("\n");
    }

    free(A);
    hipFree(d_A);
    return 0;
}